#include "hip/hip_runtime.h"
/* 
 *file name: matrix.cu
 *this program contains code to do realtively simple matrix operations
 *This was designed to work with matrices hundreds of indexes long
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "cuPrintf.cu"
#include "matrix.h"

/*
function: matrixMultiplication

this function takes in an m X n matrix h_a,
a n X k matrix h_b, and computes the matrix multiplication and 
stores the result int m x k matrix (C)
 */

__global__ void matrixMultiplication(double *h_a, double *h_b, double *h_result, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;
	if(col < k && row < m)
	{
		for(int i = 0; i < n; ++i)
		{
			sum+=h_a[row*n+i] * h_b[i*k+col];
		}
		h_result[row*k + col] = sum;
	}
}

/*
function: transposeMatrix

this function takes an rows x cols matrix inputMatrix
and transposes that matrix to outputMatrix that has dimension cols x row
i  */
__global__ void transposeMatrix(double *inputMatrix, double* outputMatrix, const unsigned int rows, const unsigned int cols)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if(idx < cols && idy < rows)
	{
		unsigned int pos = idy * cols + idx;
		unsigned int trans_pos = idx * rows + idy;
		outputMatrix[trans_pos] = inputMatrix[pos];
	}
}


/*
function: printMatrixCuda
this function prints out the matrix using CUDA
 */

__global__ void printMatrixCuda(double *C, int M, int N)
{
	cuPrintf("nothing is happening \n");
	for(int i = 0; i < M; i++)
	{
		for(int j = 0; j < N; j++)
		{
			cuPrintf("%f ", *((C+i*N) + j));
		}
		cuPrintf("\n");
	}
}

/*
function: main
 */

int main(void)
{
	printf("Hello world Matrix cuda function\n");
	double inputA[3][3];
	double inputB[3][3];
	double output[3][3];
	int m = 3;
	int n = 3;
	int k = 3;
	for(int i = 0; i < 3; ++i)
	{
		for(int j = 0; j < 3; ++j)
		{
			inputA[i][j] = j+1;
			inputB[i][j] = j+1;
		}
	}
	int BLOCK_SIZE = 16;

	unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	/*setting up how many blocks and threads we will need based on block size */
	/*mallocing temps for cuda gpu */
	double *cuda_inputA = NULL;
	double *cuda_inputB = NULL;
	double *cuda_output = NULL;

	int sizeA = sizeof(inputA);
	int sizeB = sizeof(inputB);
	int sizeO = sizeof(output);
	hipMalloc((void**) &cuda_inputA, sizeA);
	hipMalloc((void**) &cuda_inputB, sizeB);
	hipMalloc((void**) &cuda_output, sizeO);
	printf("Finished Allocating!!\n");
	/* For hipMemcpy to work, you must pass items by reference,
	 */	
	hipMemcpy(cuda_inputA, inputA, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(cuda_inputB, inputB, sizeB, hipMemcpyHostToDevice);
	//hipMemcpy(cuda_output, output, sizeO, hipMemcpyHostToDevice);
	printf("Finished Copying Cuda Values!!\n");

	printf("starting matrix Multiplication \n");
	/* the parameters <<<N, K>>> indicate that you are calling to gpu.
	   N = number of blocks 
	   k = number of threads inside blocks */
	matrixMultiplication<<<dimGrid,dimBlock>>>(cuda_inputA, cuda_inputB, cuda_output, m, n, k);
	printf("finished matrix multiplcaiton! \n");

	printf("bringing result from gpu to device....\n");
	hipMemcpy(output, cuda_output, sizeO, hipMemcpyDeviceToHost);
	printf("done!\n");
	printf("Matrix A: \n");
	/*initialize cuPrintf */

	cudaPrintfInit();
	printMatrixCuda<<<1,1>>>(cuda_inputA, m, n);
	cudaPrintfDisplay(NULL, true);
	cudaPrintfEnd();
//	printMatrix((double*)inputA, m, n);
	printf("Matrix B\n");
	cudaPrintfInit();
	printMatrixCuda<<<1,1>>>(cuda_inputB, n, k);
	cudaPrintfDisplay(NULL, true);
	cudaPrintfEnd();
//	printMatrix((double*)inputB, n, k);
	printf("Matrix Out: \n");
	cudaPrintfInit();
	printMatrixCuda<<<1,1>>>(cuda_output, m, k);
	cudaPrintfDisplay(NULL, true);
	cudaPrintfEnd();
//	printMatrix((double*)output, m, k);
	//cudaPrintfEnd();
	return 0;
}

void printMatrix(double *C, int M, int N)
{
	assert(C);
	for(int i = 0; i < M; i++)
	{
		for(int j = 0; j < N; j++)
		{
			printf("%f ", *((C+i*N) + j));
		}
		printf("\n");
	}
}

