#include "hip/hip_runtime.h"
/**
 * Author: Khoi Trinh
 * Implementation of backprop function in CUDA
 */

#include <assert.h>
#include "backprop.h"
#include "matrix.h"

// no need to include CUDA libraries since
// nvcc automatically takes care of dependencies

__global__ float* backprop_cal(float* desired_output, float* actual_output,
                               float* output_prev_layer) {
  assert(desired_output);
  assert(actual_output);
  assert(output_prev_layer);
}