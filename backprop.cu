#include "hip/hip_runtime.h"
/**
 * Author: Khoi Trinh
 * Implementation of backprop function in CUDA
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include "backprop.h"
#include "matrix.h"

#define THREAD_PER_BLOCK 64
#define REGISTERS_PER_THREAD 32
#define SHARED_MEMORY_PER_BLOCK 2048

// no need to include CUDA libraries since
// nvcc automatically takes care of dependencies

float der_error_cal_first(float desired_output, float actual_output)

{
  return actual_output - desired_output;
}

// using equation 6 in the CUDA backprop paper
// 1 block for each element in the result
__global__ void der_error_input_cal_hiddenCUDA(float* weight,
                                           int    prev_layer_node_lenth,
                                           float* cur_layer_der,
                                           int    cur_layer_node_lenth,
                                           float* prev_layer_value,
                                           float* result)
{
  int              weight_id    = blockIdx.x;
  int              cur_layer_id = threadIdx.x;
  float            sum          = 0;
  extern __shared__ float temp_sum[];
  // every element compute one multiplication then thread 0 add them up
  temp_sum[cur_layer_id] =
      weight[cur_layer_id * cur_layer_node_lenth + weight_id] *
      cur_layer_der[cur_layer_id];
  __syncthreads();
  if (threadIdx.x == 0)
    {
#pragma unroll
      for (int i = 0; i < cur_layer_node_lenth; ++i)
        {
          sum += temp_sum[i];
        }
      sum *= (1 - powf(tanhf(prev_layer_value[weight_id]), 2));
      result[weight_id] = sum;
    }
}
// expect the  to be an nx1 vectors
// and the weight to be a nxm, where n is the node
// length of the  layer and m is the node length
// of the current layer
// calculate the derivative of the pevious layer
void der_error_input_cal_hidden(float* weight,
                                int    prev_layer_node_lenth,
                                float* cur_layer_der,int    cur_layer_node_lenth,
                                float* prev_layer_value,
                                float* result)
{
  assert(weight); assert(cur_layer_der);
  assert(result);
  float* cuda_weight           = NULL;
  float* cuda_cur_layer_der    = NULL;
  float* cuda_prev_layer_value = NULL;
  float* cuda_result           = NULL;

  int size_w               = sizeof(weight);
  int size_cur_layer_der   = sizeof(cur_layer_der);
  int size_result          = sizeof(result);
  int size_cur_layer_value = sizeof(prev_layer_value);
  hipMalloc((void**)&cuda_weight, size_w);
  hipMalloc((void**)&cuda_cur_layer_der, size_cur_layer_der);
  hipMalloc((void**)&cuda_result, size_result);
  hipMalloc((void**)&cuda_prev_layer_value, size_cur_layer_value);

  hipMemcpy(cuda_weight, weight, size_w, hipMemcpyHostToDevice);
  hipMemcpy(cuda_cur_layer_der,
             cur_layer_der,
             size_cur_layer_der,
             hipMemcpyHostToDevice);
  hipMemcpy(cuda_prev_layer_value,
             prev_layer_value,
             size_cur_layer_value,
             hipMemcpyHostToDevice);

  der_error_input_cal_hiddenCUDA <<< prev_layer_node_lenth,
      cur_layer_node_lenth, cur_layer_node_lenth*sizeof(float) >>> (cuda_weight,
                               prev_layer_node_lenth,
                               cuda_cur_layer_der,cur_layer_node_lenth,
                               cuda_prev_layer_value,
                               cuda_result);
  hipMemcpy(result, cuda_result, size_result, hipMemcpyDeviceToHost);

  hipFree(cuda_cur_layer_der);
  hipFree(cuda_result);
  hipFree(cuda_prev_layer_value);
}

__global__ void backprop_calCUDA(float* der_error_value,
                                 float* prev_layer_value,int prev_layer_node_length,
                                 float* result)
{
  int weight_id_x    = blockIdx.x;
  int weight_id_y    = blockIdx.y;
  int cur_layer_id_x = threadIdx.x;
  int cur_layer_id_y = threadIdx.y;
  result[(blockDim.x * weight_id_x + cur_layer_id_x)*prev_layer_node_length+ blockDim.y * weight_id_y + cur_layer_id_y] =
            der_error_value[blockDim.y * weight_id_y + cur_layer_id_y] *
            prev_layer_value[blockDim.x * weight_id_x + cur_layer_id_x];
}

void backprop_cal(float* der_error_value,
                  int    node_length,
                  float* prev_layer_value,
                  int    prev_layer_node_length,
                  float* result)
{
  assert(der_error_value);
  assert(prev_layer_value);
  assert(result);
  float*    cuda_der_error_value  = NULL;
  float*    cuda_prev_layer_value = NULL;
  float*    cuda_result           = NULL;
  const int block_size            = 32;

  dim3 block_index(node_length / block_size, prev_layer_node_length / block_size);

  // memory allocations
  int size_der_error_value  = sizeof(der_error_value);
  int size_prev_layer_value = sizeof(prev_layer_value);
  int size_result           = sizeof(result);
  hipMalloc((void**)&cuda_der_error_value, size_der_error_value);
  hipMalloc((void**)&cuda_prev_layer_value, size_prev_layer_value);
  hipMalloc((void**)&cuda_result, size_result);

  backprop_calCUDA <<< block_index, block_size >>> (cuda_der_error_value,cuda_prev_layer_value,prev_layer_node_length,cuda_result);

  // cleanup
  hipMemcpy(result, cuda_result, size_result, hipMemcpyDeviceToHost);
  hipFree(cuda_der_error_value);
  hipFree(cuda_prev_layer_value);
  hipFree(cuda_result);
}