#include "hip/hip_runtime.h"
/* 
 *file name: matrix.cu
 *this program contains code to do realtively simple matrix operations
 *This was designed to work with matrices hundreds of indexes long
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "cuPrintf.cu"
#include "matrix.h"

/*
function: matrixMultiplication

this function takes in an m X n matrix h_a,
a n X k matrix h_b, and computes the matrix multiplication and 
stores the result int m x k matrix (C)
 */

__global__ void matrixMultiplication(float *h_a, float *h_b, float *h_result, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int sum = 0;
	if(col < k && row < m)
	{
		for(int i = 0; i < n; ++i)
		{
			sum+=h_a[row*n+i] * h_b[i*k+col];
		}
		h_result[row*k + col] = sum;
	}
}

/*
function: transposeMatrix

this function takes an rows x cols matrix inputMatrix
and transposes that matrix to outputMatrix that has dimension cols x row
i  */
__global__ void transposeMatrix(float *inputMatrix, float* outputMatrix, const unsigned int rows, const unsigned int cols)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if(idx < cols && idy < rows)
	{
		unsigned int pos = idy * cols + idx;
		unsigned int trans_pos = idx * rows + idy;
		outputMatrix[trans_pos] = inputMatrix[pos];
	}
}


/*
function: printMatrixCuda
this function prints out the matrix using CUDA
 */

__global__ void printMatrixCuda(float *C, int M, int N)
{
	cuPrintf("nothing is happening \n");
	for(int i = 0; i < M; i++)
	{
		for(int j = 0; j < N; j++)
		{
			cuPrintf("%f ", *((C+i*N) + j));
		}
		cuPrintf("\n");
	}
}

/*
function: main
 */

int main(void)
{
	printf("Hello world Matrix cuda function\n");
	float inputA[3][3];
	float inputB[3][3];
	float output[3][3];
	int m = 3;
	int n = 3;
	int k = 3;
	for(int i = 0; i < 3; ++i)
	{
		for(int j = 0; j < 3; ++j)
		{
			inputA[i][j] = j+1;
			inputB[i][j] = j+1;
		}
	}
	int BLOCK_SIZE = 16;

	unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	/*setting up how many blocks and threads we will need based on block size */
	/*mallocing temps for cuda gpu */
	float *cuda_inputA = NULL;
	float *cuda_inputB = NULL;
	float *cuda_output = NULL;

	int sizeA = sizeof(inputA);
	int sizeB = sizeof(inputB);
	int sizeO = sizeof(output);
	hipMalloc((void**) &cuda_inputA, sizeA);
	hipMalloc((void**) &cuda_inputB, sizeB);
	hipMalloc((void**) &cuda_output, sizeO);
	printf("Finished Allocating!!\n");
	/* For hipMemcpy to work, you must pass items by reference,
	 */	
	hipMemcpy(cuda_inputA, inputA, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(cuda_inputB, inputB, sizeB, hipMemcpyHostToDevice);
	//hipMemcpy(cuda_output, output, sizeO, hipMemcpyHostToDevice);
	printf("Finished Copying Cuda Values!!\n");

	printf("starting matrix Multiplication \n");
	/* the parameters <<<N, K>>> indicate that you are calling to gpu.
	   N = number of blocks 
	   k = number of threads inside blocks */
	matrixMultiplication<<<dimGrid,dimBlock>>>(cuda_inputA, cuda_inputB, cuda_output, m, n, k);
	printf("finished matrix multiplcaiton! \n");

	printf("bringing result from gpu to device....\n");
	hipMemcpy(output, cuda_output, sizeO, hipMemcpyDeviceToHost);
	printf("done!\n");
	printf("Matrix A: \n");
	/*initialize cuPrintf */

	cudaPrintfInit();
	printMatrixCuda<<<1,1>>>(cuda_inputA, m, n);
	cudaPrintfDisplay(NULL, true);
	cudaPrintfEnd();
//	printMatrix((float*)inputA, m, n);
	printf("Matrix B\n");
	cudaPrintfInit();
	printMatrixCuda<<<1,1>>>(cuda_inputB, n, k);
	cudaPrintfDisplay(NULL, true);
	cudaPrintfEnd();
//	printMatrix((float*)inputB, n, k);
	printf("Matrix Out: \n");
	cudaPrintfInit();
	printMatrixCuda<<<1,1>>>(cuda_output, m, k);
	cudaPrintfDisplay(NULL, true);
	cudaPrintfEnd();
//	printMatrix((float*)output, m, k);
	//cudaPrintfEnd();
	return 0;
}

void printMatrix(float *C, int M, int N)
{
	assert(C);
	for(int i = 0; i < M; i++)
	{
		for(int j = 0; j < N; j++)
		{
			printf("%f ", *((C+i*N) + j));
		}
		printf("\n");
	}
}

