#include "hip/hip_runtime.h"
/*
 *file name: matrix.cu
 *this program contains code to do realtively simple matrix operations
 *This was designed to work with matrices hundreds of indexes long
 */
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuPrintf.cu"
#include "matrix.h"

__global__ void matrixMultiplicationCUDA(
    float *h_a, float *h_b, float *h_result, int m, int n, int k)
{
  // calculate matrix index
  int              row = blockIdx.y * blockDim.y + threadIdx.y;
  int              col = blockIdx.x * blockDim.x + threadIdx.x;
  float            sum = 0f;
  __shared__ float temp_left_matrix[m][n];
  __shared__ float temp_right_matrix[n][k];
#pragma unroll
  for (int i = 0; i < n; ++i)
    {
      sum += h_a[row * n + i] * h_b[i * k + col];
    }
  h_result[row * k + col] = sum;
}

void matrixMultiplication(float *left_matrix,
                          float *right_matrix,
                          float *result,
                          int    row_num_left,
                          int    col_num_left,
                          int    co_num_right)
{
  assert(left_matrix);
  assert(right_matrix);
  assert(result);

  int sizeLeft  = sizeof(left_matrix);
  int sizeRight = sizeof(right_matrix);
  int sizeRes   = sizeof(result);

  // calculate correct block and thread number for indexing
  // TODO: optimize block size
  unsigned int grid_rows = (row_num_left + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_cols = (co_num_right + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3         dimGrid(grid_cols, grid_rows);
  dim3         dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  // assign and prepare memory for CUDA operation
  float *cuda_inputA = NULL;
  float *cuda_inputB = NULL;
  float *cuda_output = NULL;

  assert(hipMalloc((void **)&cuda_inputA, sizeLeft) !=
         hipErrorOutOfMemory);
  assert(hipMalloc((void **)&cuda_inputB, sizeRight) !=
         hipErrorOutOfMemory);
  assert(hipMalloc((void **)&cuda_output, sizeRes) !=
         hipErrorOutOfMemory);

  hipMemcpy(cuda_inputA, left_matrix, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(cuda_inputB, right_matrix, sizeB, hipMemcpyHostToDevice);

  matrixMultiplicationCUDA<<<dimGrid, dimBlock>>>(cuda_inputA,
                                                  cuda_inputB,
                                                  cuda_output,
                                                  row_num_left,
                                                  col_num_left,
                                                  co_num_right);

  hipMemcpy(result, cuda_output, sizeO, hipMemcpyDeviceToHost);

  // cleanup
  hipFree(cuda_inputA);
  hipFree(cuda_inputB);
  hipFree(cuda_output);
}

/*
function: transposeMatrix

this function takes an rows x cols matrix inputMatrix
and transposes that matrix to outputMatrix that has dimension cols x row
i  */
__global__ void transposeMatrix(float *            inputMatrix,
                                float *            outputMatrix,
                                const unsigned int rows,
                                const unsigned int cols)
{
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idx < cols && idy < rows)
    {
      unsigned int pos        = idy * cols + idx;
      unsigned int trans_pos  = idx * rows + idy;
      outputMatrix[trans_pos] = inputMatrix[pos];
    }
}

/*
function: printMatrixCuda
this function prints out the matrix using CUDA
 */

__global__ void printMatrixCuda(float *C, int M, int N)
{
  cuPrintf("nothing is happening \n");
  for (int i = 0; i < M; i++)
    {
      for (int j = 0; j < N; j++)
        {
          cuPrintf("%f ", *((C + i * N) + j));
        }
      cuPrintf("\n");
    }
}

/*
function: main
 */

int main(void)
{
  printf("Hello world Matrix cuda function\n");
  float inputA[3][3];
  float inputB[3][3];
  float output[3][3];
  int   m = 3;
  int   n = 3;
  int   k = 3;
  for (int i = 0; i < 3; ++i)
    {
      for (int j = 0; j < 3; ++j)
        {
          inputA[i][j] = j + 1;
          inputB[i][j] = j + 1;
        }
    }
  int BLOCK_SIZE = 16;

  /*setting up how many blocks and threads we will need based on block size */
  /*mallocing temps for cuda gpu */
  float *cuda_inputA = NULL;
  float *cuda_inputB = NULL;
  float *cuda_output = NULL;

  int sizeA = sizeof(inputA);
  int sizeB = sizeof(inputB);
  int sizeO = sizeof(output);
  hipMalloc((void **)&cuda_inputA, sizeA);
  hipMalloc((void **)&cuda_inputB, sizeB);
  hipMalloc((void **)&cuda_output, sizeO);
  printf("Finished Allocating!!\n");
  /* For hipMemcpy to work, you must pass items by reference,
   */
  hipMemcpy(cuda_inputA, inputA, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(cuda_inputB, inputB, sizeB, hipMemcpyHostToDevice);
  // hipMemcpy(cuda_output, output, sizeO, hipMemcpyHostToDevice);
  printf("Finished Copying Cuda Values!!\n");

  printf("starting matrix Multiplication \n");

  /* the parameters <<<N, K>>> indicate that you are calling to gpu.
     N = number of blocks
     k = number of threads inside blocks */
  matrixMultiplicationCUDA<<<dimGrid, dimBlock>>>(
      cuda_inputA, cuda_inputB, cuda_output, m, n, k);

  printf("finished matrix multiplcaiton! \n");

  printf("bringing result from gpu to device....\n");
  hipMemcpy(output, cuda_output, sizeO, hipMemcpyDeviceToHost);
  printf("done!\n");
  printf("Matrix A: \n");
  /*initialize cuPrintf */

  cudaPrintfInit();
  printMatrixCuda<<<1, 1>>>(cuda_inputA, m, n);
  cudaPrintfDisplay(NULL, true);
  cudaPrintfEnd();
  //	printMatrix((float*)inputA, m, n);
  printf("Matrix B\n");
  cudaPrintfInit();
  printMatrixCuda<<<1, 1>>>(cuda_inputB, n, k);
  cudaPrintfDisplay(NULL, true);
  cudaPrintfEnd();
  //	printMatrix((float*)inputB, n, k);
  printf("Matrix Out: \n");
  cudaPrintfInit();
  printMatrixCuda<<<1, 1>>>(cuda_output, m, k);
  cudaPrintfDisplay(NULL, true);
  cudaPrintfEnd();
  //	printMatrix((float*)output, m, k);
  // cudaPrintfEnd();
  return 0;
}

void printMatrix(float *C, int M, int N)
{
  assert(C);
  for (int i = 0; i < M; i++)
    {
      for (int j = 0; j < N; j++)
        {
          printf("%f ", *((C + i * N) + j));
        }
      printf("\n");
    }
}
